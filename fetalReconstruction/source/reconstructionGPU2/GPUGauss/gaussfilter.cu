#include "hip/hip_runtime.h"
/*=========================================================================
Library   : Image Registration Toolkit (IRTK)
Copyright : Imperial College, Department of Computing
Visual Information Processing (VIP), 2011 onwards
Date      : $Date: 2013-11-15 14:36:30 +0100 (Fri, 15 Nov 2013) $
Version   : $Revision: 1 $
Changes   : $Author: bkainz $

Copyright (c) 2014, Bernhard Kainz, Markus Steinberger,
Maria Murgasova, Kevin Keraudren
All rights reserved.

If you use this work for research we would very much appreciate if you cite
Bernhard Kainz, Markus Steinberger, Maria Kuklisova-Murgasova, Christina Malamateniou,
Wolfgang Wein, Thomas Torsney-Weir, Torsten Moeller, Mary Rutherford,
Joseph V. Hajnal and Daniel Rueckert:
Fast Volume Reconstruction from Motion Corrupted 2D Slices.
IEEE Transactions on Medical Imaging, in press, 2015

IRTK IS PROVIDED UNDER THE TERMS OF THIS CREATIVE
COMMONS PUBLIC LICENSE ("CCPL" OR "LICENSE"). THE WORK IS PROTECTED BY
COPYRIGHT AND/OR OTHER APPLICABLE LAW. ANY USE OF THE WORK OTHER THAN
AS AUTHORIZED UNDER THIS LICENSE OR COPYRIGHT LAW IS PROHIBITED.

BY EXERCISING ANY RIGHTS TO THE WORK PROVIDED HERE, YOU ACCEPT AND AGREE
TO BE BOUND BY THE TERMS OF THIS LICENSE. TO THE EXTENT THIS LICENSE MAY BE
CONSIDERED TO BE A CONTRACT, THE LICENSOR GRANTS YOU THE RIGHTS CONTAINED
HERE IN CONSIDERATION OF YOUR ACCEPTANCE OF SUCH TERMS AND CONDITIONS.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
list of conditions and the following disclaimer.
2. Redistributions in binary form must reproduce the above copyright notice,
this list of conditions and the following disclaimer in the documentation
and/or other materials provided with the distribution.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
=========================================================================*/

//! Functionality for performing gaussian filtering

#ifndef GAUSSFILTER_CU
#define GAUSSFILTER_CU

#include <stdio.h>
#include <npp.h>
#include "gaussFilterConvolution.cuh"
//#include "gaussfilter_kernel.cu"
#include "hip/hip_runtime_api.h"


int iDivUp(int a, int b)
{
  return (a + b - 1) / b;
  //return (a % b != 0) ? (a / b + 1) : (a / b);
}


//!/////////////////////////////////////////////////////////////////////////////
//! General Functions
//!/////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////
//! Generate 1D Gaussian convolution kernel
//! @param kernel    resulting kernel (necassary memory will be allocated)
//! @param sigma     sigma
//! @param klength   klength of the kernel
////////////////////////////////////////////////////////////////////////////////
int generateGaussianKernel(float** kernel, float sigma, int klength)
{
  // check for valid filter length
  if ((klength % 2) == 0)
  {
    fprintf(stderr, "Error: Convolution Kernel length even\n");
    return -1;
  }

  // allocate memory for kernel
  *kernel = (float*)malloc(sizeof(float) * klength);

  // sum for normalization
  float sum = 0;

  // compute kernel values
  int mid_point = (int)floor(klength/2.0f);
  for( int i = 0; i < klength; i++)
  {
    // generate value
    (*kernel)[i] = exp(-(float)abs(i-mid_point)*(float)abs(i-mid_point)/(2*sigma*sigma));

    // update sum for normalization
    sum += (*kernel)[i];
  }

  // normalize kernel
  for(int i = 0; i < klength; i++)
    (*kernel)[i] /= sum;

  return 0;
}

texture<float, hipTextureType1D, hipReadModeElementType> gaussKernelTex_;


template<int klength>
__global__ void GaussXKernel(hipSurfaceObject_t in, hipSurfaceObject_t out)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z;

  float v = surf2DLayeredread<float>(in, x*4, y, z, hipBoundaryModeClamp);
  
  if(v != -1)
  {
    v = v * tex1Dfetch(gaussKernelTex_, 0);
 
    #pragma unroll
    for(int i = 1; i < (klength+1)/2; ++i)
      v = v + tex1Dfetch(gaussKernelTex_, i) * (max(0.0f,surf2DLayeredread<float>(in, (x + i)*4, y, z, hipBoundaryModeClamp)) + 
                                                max(0.0f,surf2DLayeredread<float>(in, (x - i)*4, y, z, hipBoundaryModeClamp)) );
  }

  surf2DLayeredwrite(v, out, x*4, y, z, hipBoundaryModeZero);
}


__global__ void GaussXKernelGeneral(int klength, hipSurfaceObject_t in, hipSurfaceObject_t out)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z;

  float v = surf2DLayeredread<float>(in, x*4, y, z, hipBoundaryModeClamp);
  
  if(v != -1)
  {
    v = v * tex1Dfetch(gaussKernelTex_, 0);

    for(int i = 1; i < (klength+1)/2; ++i)
      v = v + tex1Dfetch(gaussKernelTex_, i) * (max(0.0f,surf2DLayeredread<float>(in, (x + i)*4, y, z, hipBoundaryModeClamp)) + 
                                                max(0.0f,surf2DLayeredread<float>(in, (x - i)*4, y, z, hipBoundaryModeClamp)) );
  }
 
   surf2DLayeredwrite(v, out, x*4, y, z, hipBoundaryModeZero);
}

template<int klength>
__global__ void GaussYKernel(hipSurfaceObject_t in, hipSurfaceObject_t out)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z;

  float v = surf2DLayeredread<float>(in, x*4, y, z, hipBoundaryModeClamp);
  
  if(v != -1)
  {
    v = v * tex1Dfetch(gaussKernelTex_, 0);
    #pragma unroll
    for(int i = 1; i < (klength+1)/2; ++i)
      v = v + tex1Dfetch(gaussKernelTex_, i) * (max(0.0f,surf2DLayeredread<float>(in, x*4, y + i, z, hipBoundaryModeClamp)) + 
                                                max(0.0f,surf2DLayeredread<float>(in, x*4, y - i, z, hipBoundaryModeClamp)) );
  }
 
  surf2DLayeredwrite(v, out, x*4, y, z, hipBoundaryModeZero);
}

__global__ void GaussYKernelGeneral(int klength, hipSurfaceObject_t in, hipSurfaceObject_t out)
{ 
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z;

  float v = surf2DLayeredread<float>(in, x*4, y, z, hipBoundaryModeClamp);
  
  if(v != -1)
  {
    v = v * tex1Dfetch(gaussKernelTex_, 0);
  
    for(int i = 1; i < (klength+1)/2; ++i)
      v = v + tex1Dfetch(gaussKernelTex_, i) * (max(0.0f,surf2DLayeredread<float>(in, x*4, y + i, z, hipBoundaryModeClamp)) + 
                                                max(0.0f,surf2DLayeredread<float>(in, x*4, y - i, z, hipBoundaryModeClamp)) );
  }
  surf2DLayeredwrite(v, out, x*4, y, z, hipBoundaryModeZero);
}


////////////////////////////////////////////////////////////////////////////////
//! Performes optimized gaussian filtering of a stack of image (x,y direction
//! while slices are stacked up along z
//! @param input          pointer to input image stack
//! @param output         pointer to output image stack
//! @param temp           pointer to temp image stack
//! @param width          width of the image
//! @param height         height of the image
//! @param slices         num slices
//! @param pitchX/Y       image sizes
//! @param num_ch         number of channels in the image
//! @param sigma          sigma parameter to construct kernel
////////////////////////////////////////////////////////////////////////////////
int FilterGaussStack(hipSurfaceObject_t input, hipSurfaceObject_t output, hipSurfaceObject_t temp, 
                 unsigned int width, unsigned int height, unsigned int slices, float sigma)
{
  int ret = 0;

  //determine filter length
  int klength = max(min((int)(sigma*5),MAX_LENGTH_SK),7);
  klength -= 1-klength%2;

  int dev;
  hipGetDevice(&dev);

  static int lastKLength[128] =      {-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1};
  static float lastsigma[128] =      { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
  static float* d_GaussKoeffs[128] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
  if(lastKLength[dev] != klength || lastsigma[dev] != sigma)
  {
    if(lastKLength[dev] != klength)
    {
      if(d_GaussKoeffs[dev] != 0)
        hipFree(d_GaussKoeffs[dev]);
      hipMalloc(&d_GaussKoeffs[dev], sizeof(float)*(klength+1)/2);
    }

    // generate kernel
    float* kernel = NULL;
    ret = generateGaussianKernel(&kernel, sigma, klength);
    if (ret)
    {
      fprintf(stderr, "Error in CUDA FilterGaussStack(): Could not generate Kernel\n");
      return ret;
    }

    hipMemcpy(d_GaussKoeffs[dev], kernel + klength/2, (klength+1)/2*sizeof(float), hipMemcpyHostToDevice);

    free(kernel);

    

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    checkCudaErrors(hipBindTexture(0, gaussKernelTex_, d_GaussKoeffs[dev], hipCreateChannelDesc<float>(),(klength+1)/2*sizeof(float)));

    gaussKernelTex_.addressMode[0] = hipAddressModeClamp;
    gaussKernelTex_.filterMode = hipFilterModePoint;
    gaussKernelTex_.normalized = false;

    lastsigma[dev] = sigma;
    lastKLength[dev] = klength;
  }


  //filter (with optimizations for special cases)
  const int blockSize1 = 32;
  const int blockSize2 = 32;

  dim3 blockx(blockSize1, blockSize2);
  dim3 gridx(iDivUp(width, blockSize1), iDivUp(height, blockSize2), slices);
  dim3 blocky(blockSize2, blockSize1);
  dim3 gridy(iDivUp(width, blockSize2), iDivUp(height, blockSize1), slices);

  switch(klength)
  {
  case 7:
    GaussXKernel<7><<<gridx, blockx>>>(input, temp);
    GaussYKernel<7><<<gridy, blocky>>>(temp, output);
    break;
  case 9:
    GaussXKernel<9><<<gridx, blockx>>>(input, temp);
    GaussYKernel<9><<<gridy, blocky>>>(temp, output);
    break;
  case 11:
    GaussXKernel<11><<<gridx, blockx>>>(input, temp);
    GaussYKernel<11><<<gridy, blocky>>>(temp, output);
    break;
  case 13:
    GaussXKernel<13><<<gridx, blockx>>>(input, temp);
    GaussYKernel<14><<<gridy, blocky>>>(temp, output);
    break;
  case 15:
    GaussXKernel<15><<<gridx, blockx>>>(input, temp);
    GaussYKernel<15><<<gridy, blocky>>>(temp, output);
    break;
  default:
    GaussXKernelGeneral<<<gridx, blockx>>>(klength, input, temp);
    GaussYKernelGeneral<<<gridy, blocky>>>(klength, temp, output);
    break;
  }
  return ret;
}


#endif // GAUSSFILTER_CU
